#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <malloc.h>
#include <string.h>
#include <math.h>

// perform: weights = weights - learning_rate * delta_weights
//          delta_weights = (1/data_count) * delta_mat * in_mat^T
//  delta_mat: matrix of delta [row,column] = [neuron, sample]
//  in_mat   : matrix of input [row,column] = [data_dim, sample]
//  weights  : matrix of weight[row,column] = [neuron, data_dim]
__global__ void kernel_fc_calc_weights_diff_by_bp(int data_count, int neuron_count, int in_data_dim, float learning_rate, float *delta_mat, float *in_mat, float *weights, float *weights_diff)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= neuron_count * in_data_dim) return;

	int neuron_idx = index % neuron_count;
	int target_dim = index / neuron_count;

	float *delta_begin = &delta_mat[neuron_idx*data_count];
	float *in_begin    = &in_mat[target_dim*data_count];

	float sum = 0;
	for (int data_i = 0; data_i < data_count; data_i++)
	{
		float delta = delta_begin[data_i];
		float in = in_begin[data_i];

		sum += delta * in;
	}

	weights_diff[neuron_idx * in_data_dim + target_dim] = learning_rate * sum / (data_count);
}

// call cuda kernel
//  delta_mat: matrix of delta [row,column] = [neuron, sample]:   must be a single array
//  in_mat   : matrix of input [row,column] = [data_dim, sample]: must be a single array
//  weights  : matrix of weight[row,column] = [neuron, data_dim]: must be a single array
//  o_weights_diff: output matrix of weight[row, column] = [neuron, data_dim] : must be a single array
void cuda_fc_calc_weights_diff_by_bp(int data_count, int neuron_count, int in_data_dim, float learning_rate, float *delta_mat, float *in_data_mat, float *weights, float *o_weights_diff)
{
	float *k_delta_mat;
	float *k_in_data_mat;
	float *k_weights, *k_weights_diff;

//#define CHECK_RESULT


	hipMalloc(&k_delta_mat, neuron_count * data_count * sizeof(float));
	hipMalloc(&k_in_data_mat, data_count   * in_data_dim * sizeof(float));
	hipMalloc(&k_weights, neuron_count * in_data_dim * sizeof(float));
	hipMalloc(&k_weights_diff, neuron_count * in_data_dim * sizeof(float));

	hipMemcpy(k_delta_mat, delta_mat, neuron_count * data_count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(k_in_data_mat, in_data_mat, data_count   * in_data_dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(k_weights, weights, neuron_count * in_data_dim * sizeof(float), hipMemcpyHostToDevice);

	int thread_count = neuron_count * in_data_dim;

	// Perform kernel (256 threads in each block)
	kernel_fc_calc_weights_diff_by_bp << <(thread_count + 255) / 256, 256 >> > (data_count, neuron_count, in_data_dim, learning_rate, k_delta_mat, k_in_data_mat, k_weights, k_weights_diff);

	hipMemcpy(o_weights_diff, k_weights_diff, neuron_count * in_data_dim * sizeof(float), hipMemcpyDeviceToHost);

#ifdef CHECK_RESULT
	// validate
	// weights[0] (=weights[neuron=0][dim=0]) == prev_weights[0] - learning_rate / data_count * sum(delta[neuron=0] * in[dim=0]) for sample
	for (size_t neuron = 0; neuron < neuron_count; neuron++)
	{
		for (size_t dim = 0; dim < in_data_dim; dim++)
		{
			float sum = 0;
			float cur_v = o_weights_diff[neuron * in_data_dim + dim] / 10000;
			for (size_t i = 0; i < data_count; i++)
			{
				sum += delta_mat[neuron * data_count + i] * in_data_mat[dim * data_count + i];
			}
			float desired = learning_rate * sum / data_count;

			// �w�K���i�ނ�loss�̒l���������Ȃ��ďd�݂�GPU�ł͍X�V�ł��Ȃ��Ȃ�i���x���j
			if (fabs(desired - cur_v) > 0.000001)
				printf("avg=%f, desired_weight_diff=%f, new_weight_diff=%f\n", sum / data_count, desired, cur_v);
		}
	}
#endif


}
